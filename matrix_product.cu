
#include <hip/hip_runtime.h>
#include <iostream>

using namespace std;

__global__ void matrixProduct(float *matrix_1, float *matrix_2, float *result_matrix, int matrix_size) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < matrix_size * matrix_size) {
        int row = i / matrix_size;
        int col = i % matrix_size;
        *(result_matrix + matrix_size * row + col) = 0.0;
        for (int k = 0; k < matrix_size; k++) {
            *(result_matrix + matrix_size * row + col) += *(matrix_1 + matrix_size * row + k) * *(matrix_2 + matrix_size * k + col);
        }
    }
}

int main(void) {
    cout << "Defining variables & allocating memory in HOST..." << endl;
    size_t matrix_size = 10240;
    size_t matrix_entries = matrix_size * matrix_size;
    size_t n_bytes = sizeof(float) * matrix_entries;
    float *matrix_1 = (float *) malloc(n_bytes);
    float *matrix_2 = (float *) malloc(n_bytes);
    float *result_matrix = (float *) malloc(n_bytes);

    cout << "Defining variables & allocating memory in DEVICE..." << endl;
    float *device_matrix_1;
    float *device_matrix_2;
    float *device_result_matrix;
    hipMalloc((void **) &device_matrix_1, n_bytes);
    hipMalloc((void **) &device_matrix_2, n_bytes);
    hipMalloc((void **) &device_result_matrix, n_bytes);

    cout << "Initializing variables in HOST..." << endl;
    for (int i = 0; i < matrix_size; i++) {
        for (int j = 0; j < matrix_size; j++) {
            *(matrix_1 + matrix_size * i + j) = (float) i / (j + 1);
            *(matrix_2 + matrix_size * i + j) = (float) (i + 5) / (j + 8);
        }
    }

    cout << "Copying HOST variables to DEVICE variables..." << endl;
    hipMemcpy(device_matrix_1, matrix_1, n_bytes, hipMemcpyHostToDevice);
    hipMemcpy(device_matrix_2, matrix_2, n_bytes, hipMemcpyHostToDevice);

    cout << "Defining & calling kernel..." << endl;
    int block_size = 1024;
    int blocks_count = (matrix_entries / block_size) + (matrix_entries % block_size != 0);
    matrixProduct<<<blocks_count, block_size>>>(device_matrix_1, device_matrix_2, device_result_matrix, matrix_size);

    cout << "Copying DEVICE variables to HOST variables..." << endl;
    hipMemcpy(result_matrix, device_result_matrix, n_bytes, hipMemcpyDeviceToHost);

    cout << "Checking result..." << endl;
    bool pass = true;
    for (int i = 0; i < matrix_size; i++) {
        for (int j = 0; j < matrix_size; j++) {
            float product = 0.0;
            for (int k = 0; k < matrix_size; k++) {
                product += *(matrix_1 + matrix_size * i + k) * *(matrix_2 + matrix_size * k + j);
            }
            if (abs(*(result_matrix + matrix_size * i + j) - product) > 0.1) {
                cout << "result_matrix[" << i << "][" << j << "] = " << *(result_matrix + matrix_size * i + j) << " != " << product << endl;
                pass = false;
                break;
            }
        }
        if (!pass)
            break;
    }
    if (pass)
        cout << "Passed!" << endl;
    else
        cout << "Failed!" << endl;

    cout << "Freeing resources..." << endl;
    free(matrix_1);
    free(matrix_2);
    free(result_matrix);
    hipFree(device_matrix_1);
    hipFree(device_matrix_2);
    hipFree(device_result_matrix);

    return 0;
}
